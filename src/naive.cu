#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include "types.cuh"
#include "helpers.cuh"

namespace cg = cooperative_groups;

namespace cross
{

/**
 * This kernel is a reimplementation of the original naive cross_corr kernel
 * The kernel receives reference subregions, each in row major order all stacked one after another
 * into a single array "ref". "deformed" contains corresponding subregions from "batch_size" of the deformed  pictures
 * which are to be cross-correlated with the reference subregions. All subregions are in row major order, first
 * all subregions of the first deformed image, then all subregions of the second deformed image up to the "batch_size"th
 * deformed image. Number of subregions from the reference and all the deformed images is the same.
 * The input arrays ref and deformed contain only the subregions themselfs, and we must
 * clamp the computation to use only the overlapping parts.
 *
 * For each subregion we search an area of the size "search_size" for cross-correlation maximum.
 * The whole strip of deformed subregions is partitioned into a 16x16 CUDA blocks,
 * where each thread computes one possible shift of the reference image.
 * Output contains an an array of "search_size" results in row major order
 * corresponding to the result of cross correlation for each position in the search area.
 *
 * The memory access patterns are not ideal. Due to the 16x16 size of each block,
 * each half of the warp accesses different row of the "picture", most likely leading to two 128 byte
 * global memory accesses. The implementation also does not use shared memory in any way.
 */
template<typename T, typename RES>
__global__ void cross_corr_naive_original(
    const T* __restrict__ ref,
    const T* __restrict__ deformed,
    RES* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size

) {
    cg::thread_block ctb = cg::this_thread_block();

    // Coordinates in the whole strip of deformed subregions
    unsigned int def_strip_x = ctb.group_index().x * ctb.group_dim().x + ctb.thread_index().x;
    unsigned int def_strip_y = ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y;

    unsigned int region_idx = def_strip_x / search_size.x;

    if (region_idx >= subregions_per_pic || def_strip_y >= search_size.y) {
        return;
    }

    // Position of the centre of the subregion
    dsize2_t in_region_pos = { def_strip_x % search_size.x, def_strip_y };
    dsize_t ref_idx = region_idx % subregions_per_pic;
    dsize2_t half_size = (search_size - 1) / 2;

    vec2<int> shift = {(int)in_region_pos.x - (int)half_size.x, (int)in_region_pos.y - (int)half_size.y};

    ref += ref_idx * subregion_size.area();
    deformed += region_idx * subregion_size.area();
    out += region_idx * search_size.area();

    for (dsize_t i = 0; i < batch_size; ++i) {
        // The code is different from the original as here we are sliding the
        // deformed region over the reference region, whereas the original
        // did it the other way, which is incorrect in my opinion
        // or at least inconsistent with the text of the thesis
        // where it is defined as reference * deformed
        // and the algorithm clearly states that this means sliding the deformed
        //
        // The results also now match the results of matlab xcorr2
        dsize_t x_ref_start = max(shift.x, 0);
        dsize_t x_ref_end = min(subregion_size.x + shift.x, subregion_size.x);
        dsize_t y_ref_start = max(shift.y, 0);
        dsize_t y_ref_end = min(subregion_size.y + shift.y, subregion_size.y);

        RES sum = 0;
        for (dsize_t y_ref = y_ref_start; y_ref < y_ref_end; ++y_ref) {
            for (dsize_t x_ref = x_ref_start; x_ref < x_ref_end; ++x_ref) {
                // If deformed is shifted by -10, the we are starting from [0,0] in ref
                // and need to start from [10,10] in deformed, as there are 10
                // values to the left and on top outside the reference matrix
                int x_shifted = x_ref - shift.x;
                int y_shifted = y_ref - shift.y;

                sum += deformed[y_shifted * subregion_size.x + x_shifted] * ref[y_ref * subregion_size.x + x_ref];
            }
        }

        out[in_region_pos.linear_idx(search_size.x)] = sum;

        deformed += subregions_per_pic * subregion_size.area();
        out += subregions_per_pic * search_size.area();
    }
}

/*
TODO: Try loading the row of reference subregion to shared memory
then do the multiplication of each ref row element with each deformed row element,
either materializing this matrix or computing it during a sum.

As each pair of ref and deformed are multiplied only in a signle shift, it may be better to just
do it during the sum computation.

So with single dimensional blocks, we just load the ref row (or part of it) and
each thread goes linearily through it's deformed row and computes the
multiplication of each element with the corresponding ref element.

Threads iterate this computation over the ref elemens to use shared memory broadcast.
So that each thread in a warp accesses the same ref element and multiplies it with the elements
in its deformed row.

As each thread in a warp will be processing a successive elements of the deformed row,
we could do thread shuffle to reduce the accesses to global memory.




TODO: Try using the whole warp for computation of a single shift
*/

template<typename T, typename RES>
void run_cross_corr_naive_original(
    const T* __restrict__ ref,
    const T* __restrict__ deformed,
    RES* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size
) {
    dim3 num_threads(16, 16);
    dim3 num_blocks(
        div_up(search_size.x * subregions_per_pic, num_threads.x),
        div_up(search_size.y, num_threads.y)
    );

    cross_corr_naive_original<<<num_blocks, num_threads>>>(
        ref,
        deformed,
        out,
        subregion_size,
        search_size,
        subregions_per_pic,
        batch_size
    );
}

template void run_cross_corr_naive_original<int, int>(
    const int* __restrict__ ref,
    const int* __restrict__ deformed,
    int* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size
);

template void run_cross_corr_naive_original<float, float>(
    const float* __restrict__ ref,
    const float* __restrict__ deformed,
    float* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size
);

template void run_cross_corr_naive_original<double, double>(
    const double* __restrict__ ref,
    const double* __restrict__ deformed,
    double* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size
);

}
