#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include "types.cuh"
#include "helpers.cuh"
#include "shared_mem.cuh"
#include "matrix.cuh"
#include "clamps.cuh"

namespace cg = cooperative_groups;

namespace cross
{


/*
TODO: Try loading the row of reference subregion to shared memory
then do the multiplication of each ref row element with each deformed row element,
either materializing this matrix or computing it during a sum.

As each pair of ref and deformed are multiplied only in a signle shift, it may be better to just
do it during the sum computation.

So with single dimensional blocks, we just load the ref row (or part of it) and
each thread goes linearily through it's deformed row and computes the
multiplication of each element with the corresponding ref element.

Threads iterate this computation over the ref elemens to use shared memory broadcast.
So that each thread in a warp accesses the same ref element and multiplies it with the elements
in its deformed row.

As each thread in a warp will be processing a successive elements of the deformed row,
we could do thread shuffle to reduce the accesses to global memory.


TODO: Try using the whole warp for computation of a single shift
*/

template<typename T>
__device__ dsize_t load_row_chunk(const cg::thread_block& ctb, T* dst, const T* src, dsize_t row_start, dsize_t row_size, dsize_t chunk_start, dsize_t chunk_size) {
    dsize_t copy_size = min(row_size - chunk_start, chunk_size);
    for (; chunk_start + ctb.thread_index().x < copy_size; chunk_start += ctb.size()) {
        dst[ctb.thread_index().x] = src[row_start + chunk_start + ctb.thread_index().x];
    }
    return copy_size;
}

/** Limited to one reference subregion and one deformed subregion
 *
 * This is just a first simple implementation, future implementations will build on top
 * of this and optimize it
 * Each block processes one row of the ref matrix
 *
 * Each block loads a <block_size> number of elements from a row of the reference subregion and
 * then goes through the deformed subregion, loading it in <block_size> chunks into shared memory.
 *
 * Each
 */
// template<typename T, typename RES>
// __global__ void ccn_shared_mem_rows(
//     const T* __restrict__ ref,
//     const T* __restrict__ deformed,
//     RES* __restrict__ out,
//     dsize2_t subregion_size,
//     dsize2_t search_size
// ) {
//     cg::thread_block ctb = cg::this_thread_block();

//     const dsize_t ref_chunk_size = ctb.size();
//     const dsize_t def_chunk_size = ctb.size();

//     // Split shared memory
//     extern __shared__ T shared[];
//     T* ref_s = shared;
//     T* def_s = ref_s + ref_chunk_size;
//     RES* shift_sums = def_s + 2*def_chunk_size + ref_chunk_size;

//     dsize2_t half_search_size = (search_size - 1) / 2;

//     const dsize_t ref_row = ctb.group_index();
//     const dsize_t ref_row_start = ref_row * subregion_size.x;

//     for (dsize_t ref_chunk_start = 0; ref_chunk_start < subregion_size.x; ref_chunk_start += ref_chunk_size) {
//         // Load part of the ref row into shared memory
//         load_row_chunk(ctb, ref_s, ref, ref_row_start, subregion_size.x, ref_chunk_start, ref_chunk_size);
//         ctb.sync();

//         // Load ref chunk into thread registers
//         // T ref_val = ref_chunk_start + ctb.thread_index().x < subregion_size.x ?
//         //     ref[ref_row_start + ref_chunk_start + ctb.thread_index().x] :
//         //     0;

//         // First deformed row to compute cc with
//         const dsize_t def_begin_row = max((int)ref_row - (int)half_search_size.y, 0);
//         const dsize_t def_end_row = min(ref_row + half_search_size.y, subregion_size.y);
//         for (dsize_t def_row = def_begin_row; def_row < def_end_row; ++def_row) {

//             const dsize_t def_row_start = def_row * subregion_size.x;
//             // For given ref row chunk, we only need to load parts of the def row
//             // that are at most search_size.x before or after
//             const dsize_t def_row_part_start = max(def_row_start, (int)def_row_start + ref_chunk_start - half_search_size.x);
//             const dsize_t def_row_part_end = min(def_row_start + subregion_size.x, def_row_part_start + ref_chunk_size + search_size.x);
//             const dsize_t def_row_part_size = def_row_part_end - def_row_part_start;

//             for (dsize_t def_chunk_start = 0; def_chunk_start < def_row_part_size; def_chunk_start += def_chunk_size) {
//                 load_row_chunk(ctb, def_s, deformed, def_row_part_start, def_row_part_size, def_chunk_start, def_chunk_size);
//                 ctb.sync();



//                 auto ctw = cg::tiled_partition<32>(ctb);
//                 auto num_warps = ctw.meta_group_size();
//                 RES shift_sum = 0;
//                 for (dsize_t shift = 0; shift < 32; ++shift) {

//                 }

//                 // TODO: Add the number of threads as param


//             }
//         }
//     }
// }

/** Two part ring buffer
 *
 * This buffer holds two submatrices, which should be neighbours in the source matrix.
 * When loading, the older submatrix is overwritten
 *
 * matrices are stored in row major order and should be continuous in the x axis
 *
 * Currently this datastructure is designed to work with blocks with 32x32 threads.
 */
// template<typename T>
// class ring_buffer {
// public:
//     __host__ __device__ ring_buffer(T* data, dsize2_t size)
//         :data_(data), size_(size)
//     {

//     }

//     __device__ void load(const cg::thread_block& ctb, const T* src, dsize2_t pos) {

//     }

//     __host__ __device__ T operator[](dsize2_t pos) {

//     }
// private:
//     T* data_;
//     dsize2_t pos_;
//     dsize2_t size_;
// };

dsize_t __device__ get_x_def_block_start(dsize_t block_start_x, int max_block_shift_x) {
    return max((int)block_start_x - max_block_shift_x, 0);
}

/**
 * This kernel takes advantage of the access pattern in the original implementation
 * where threads in the same thread block read 15/16 of the same values as different
 * threads from the same thread block in previous step. In the original implementation
 * it is done in the form of 16x16 matrix, which shares 15 columns with the previous matrix.
 *
 * Here we use 1D thread block, so each step would share all but one value from ref and def
 * matrices from the previous step. We use this to preload the next thread block worth of
 * data in one step to shared memory and use this to prevent repeated accesses to global memory,
 * even if those might have been cached.
 *
 * Whereas all threads in a single block access the same <thread_block_size> values from ref,
 * they start at <thread_block_size> different values in the corresponding def row and read
 * another <thread_block_size> values from their initial value. This means that each loop,
 * the whole block accesses <thread_block_size> values from ref but 2x<thread_block_size> values
 * from def. Important note here is that half of the def values is accessed in the next loop too,
 * so we need something like a ring buffer where each loop we load just half of the 2x<thread_block_size>
 * values, allowing us to do this load in one step.
 *
 * TODO: The following is no longer true
 * All threads read from the same <thread_block_size> values of ref
 * mostly using broadcast, but multiplying them with different but continuous
 * values from def matrix
 * If perfectly aligned, they would all do broadcast from ref and then each read
 * different value from def, where consecutive threads would read consecutive values
 */
template<typename T, typename RES>
__global__ void ccn_ring_buffer_row(
    const T* __restrict__ ref,
    const T* __restrict__ def,
    RES* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size
) {
    // TODO: Check block-wide shuffle down from cooperative groups


    auto ref_mat = matrix_slice<const T>::from_position_size(
        dsize2_t{0,0},
        subregion_size,
        subregion_size.x,
        ref
    );

    auto def_mat = matrix_slice<const T>::from_position_size(
        dsize2_t{0,0},
        subregion_size,
        subregion_size.x,
        def
    );

    cg::thread_block ctb = cg::this_thread_block();

    // Each offset between ref and def is computed by a single thread
    // so each thread corresponds to a single value in the resulting matrix
    // same as the original
    // As this kernel is simplified and only has 1D thread blocks, each thread block
    // works in a single row
    dsize2_t result_pos{
        ctb.group_index().x * ctb.group_dim().x + ctb.thread_index().x,
        ctb.group_index().y * ctb.group_dim().y
    };

    // Index of the leftmost thread in the block
    unsigned int block_res_x = ctb.group_index().x * ctb.group_dim().x;

    T* shared = shared_memory_proxy<T>();

    shared_mem_buffer<T> ref_s = shared_mem_buffer<T>::allocate(&shared, ctb.size() * 2);
    shared_mem_buffer<T> def_s = shared_mem_buffer<T>::allocate(&shared, ctb.size() * 2);

    dsize2_t half_size = (search_size - 1) / 2;
    // As thread indicies are only positive, and half of the resulting matrix
    // represents negative shifts, we need to shift by half the search size to left
    vec2<int> shift{(int)result_pos.x - (int)half_size.x, (int)result_pos.y - (int)half_size.y};

    // Minimal shift computed by any thread in the same block
    // As all threads share shift in the y axis, and value of shift along the x axis
    // is dependent on threadId in x axis, thread 0 will always have the lowest shift
    vec2<int> min_block_shift{(int)block_res_x - (int)half_size.x, (int)result_pos.y - (int)half_size.y};

    // Similarly with the min shift, max shift will always be computed by the
    // last thread in the current block
    // ctb.size() - 1 is index of the last thread in this block
    vec2<int> max_block_shift{min_block_shift.x + (int)ctb.size() - 1, min_block_shift.y};

    // Slice of the ref matrix which overlaps with the deformed matrix shifted by
    // <shift> and thus needs to be computed by the current thread
    // Specific for current thread
    auto ref_slice = matrix_slice<const T>::from_positions(
        dsize2_t{clamp_to_nonnegative(shift.x), clamp_to_nonnegative(shift.y)},
        dsize2_t{
            clamp_down(subregion_size.x + shift.x, subregion_size.x),
            clamp_down(subregion_size.y + shift.y, subregion_size.y)
        },
        subregion_size.x,
        ref
    );

    // Slice of ref matrix containing union of all ref_slices for all threads within current thread block
    // Basically part of the ref matrix accessed by any thread from the current thread block
    auto ref_slice_block = matrix_slice<const T>::from_positions(
        dsize2_t{
            clamp_to_nonnegative(min_block_shift.x),
            ref_slice.begin_y_src_idx()},
        dsize2_t{
            clamp_down(subregion_size.x + max_block_shift.x, subregion_size.x),
            ref_slice.end_y_src_idx()
        },
        subregion_size.x,
        ref
    );


    RES sum = 0;
    for (dsize_t row = 0; row < ref_slice_block.size().y; ++row) {
        // As we are only going through the parts of ref matrix that overlap def matrix
        // any shifted_. indicies should be valid
        int def_y = ref_slice_block.begin_y_src_idx() + row - shift.y;
        auto def_row = def_mat.row(def_y);
        // TODO: Most things are shared between rows, so no need to recompute them each time
        auto ref_buffer = make_row_ring_buffer<2>(
            ctb,
            ref_slice_block.row(row),
            std::move(ref_s)
        );

        // When def is shifted by -5 on x, and we access item 0 in ref, we want item 5 in def,
        // so ref.x - shift.x gives us the index in def matrix
        // Preloads the first 2 parts of the buffer
        dsize_t def_row_start_idx = max((int)ref_buffer.start_offset() - max_block_shift.x, 0);
        auto def_buffer = make_row_ring_buffer<2>(
            ctb,
            def_row.subslice(
                min(ref_slice_block.size().x, subregion_size.x - def_row_start_idx),
                def_row_start_idx
            ),
            std::move(def_s)
        );

        // Relative offsets of the two buffers stay the same during the whole row processing
        int def_buffer_thread_offset = (int)ref_buffer.start_offset() - (int)def_buffer.start_offset() - shift.x;
        do {
            // Sync after load
            ctb.sync();

            // Indicies in the ref_s buffer which should be processed by the current thread
            dsize_t ref_buffer_thread_start = max((int)ref_slice.begin_x_src_idx() - (int)ref_buffer.start_offset(), 0);
            dsize_t ref_buffer_thread_end = min(
                (int)ref_slice.end_x_src_idx() - (int)ref_buffer.start_offset(),
                min(
                    ref_buffer.num_loaded(),
                    (int)def_buffer.num_loaded() - ((int)ref_buffer_thread_start + (int)def_buffer_thread_offset)
                )
            );



            // if (ctb.group_index().y == 0 && ctb.thread_rank() == 0) {
            //     printf("Block: %u, Shift: %d, Start: %u, End: %u, Offset: %d, Ref start: %u, Def start: %u\n", ctb.group_index().x,  shift.x, ref_buffer_thread_start, ref_buffer_thread_end, def_buffer_thread_offset, ref_buffer.start_offset(), def_buffer.start_offset());
            // }

            for (dsize_t ref_buffer_index = ref_buffer_thread_start; ref_buffer_index < ref_buffer_thread_end; ++ref_buffer_index) {

                sum += ref_buffer[ref_buffer_index] * def_buffer[(int)ref_buffer_index + def_buffer_thread_offset];
            }



            // Sync after computation
            ctb.sync();
        } while (ref_buffer.load_next(ctb) && def_buffer.load_next(ctb));
    }

    if (result_pos.x < search_size.x) {
        out[result_pos.linear_idx(search_size.x)] = sum;
    }
}

/** Ring buffers
 *
 * The original implementation does pretty chaotic memory accesses and code divergence due to the x_ref and y_ref clamping
 *
 * We would like to manipulate the original implementation so that the whole 16x16 threadblock accesses two 16x16
 * submatrices each cycle and does elementwise multiplication, where each thread gets
 * one result element and adds it to its private sum
 *
 * The following cycle the block accesses two 16x16 matrices, which each share
 * 16x15 submatrix with the previous matrix. We could basically just do a ring buffer,
 * where the first warp adds to the ringbuffer each cycle, overwriting the oldest data
 *

 */
// template<typename T, typename RES>
// __global__ void ccn_ring_buffer_matrix(
//     const T* __restrict__ ref,
//     const T* __restrict__ deformed,
//     RES* __restrict__ out,
//     dsize2_t subregion_size,
//     dsize2_t search_size
// ) {
//     // TODO: Check block-wide shuffle down from cooperative groups


//     cg::thread_block ctb = cg::this_thread_block();

//     const dsize_t buffer_size = ctb.size() * 2;

//     extern __shared__ T shared[];

//     //
//     for (dsize_t )
// }




template<typename T, typename RES>
void run_ccn_ring_buffer_row(
    const T* __restrict__ ref,
    const T* __restrict__ deformed,
    RES* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t threads_per_block
) {
    // TODO: Launch multiple kernels
    dim3 num_blocks(
        div_up(search_size.x, threads_per_block),
        search_size.y
    );

    std::cout << "[" << num_blocks.x << ", " << num_blocks.y << "]\n";

    dsize_t shared_mem_size = 4 * threads_per_block * sizeof(T);

    ccn_ring_buffer_row<<<num_blocks, threads_per_block, shared_mem_size>>>(
        ref,
        deformed,
        out,
        subregion_size,
        search_size
    );
}

template void run_ccn_ring_buffer_row<int, int>(
    const int* __restrict__ ref,
    const int* __restrict__ deformed,
    int* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t threads_per_block
);

template void run_ccn_ring_buffer_row<float, float>(
    const float* __restrict__ ref,
    const float* __restrict__ deformed,
    float* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t threads_per_block
);

template void run_ccn_ring_buffer_row<double, double>(
    const double* __restrict__ ref,
    const double* __restrict__ deformed,
    double* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t threads_per_block
);

}
