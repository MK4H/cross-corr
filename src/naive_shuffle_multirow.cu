#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"
#include "shared_mem.cuh"

#include "row_distribution.cuh"

namespace cg = cooperative_groups;

namespace cross {

constexpr unsigned int warp_size = 32;

/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    vec2<int> warp_max_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        vec2<int> warp_max_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    )   : left(left), right(right), out(out), warp_right_start(warp_right_start),
    warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), warp_max_shift(warp_max_shift),
    output_pos(output_pos), matrix_size(matrix_size), search_size(search_size)
    {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    vec2<int> warp_max_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<dsize_t NUM_RIGHT_ROWS, bool REVERSE_OUTPUT, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void compute_row_group(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    dsize_t warp_y_right_start,
    int y_shift,
    RES* __restrict__ res
) {
    dsize_t warp_y_left = warp_y_right_start + y_shift;
    const T* left_row = args.left + warp_y_left * args.matrix_size.x;

    const dsize_t first_right_row_offset = warp_y_right_start * args.matrix_size.x;
    const T* first_right_row = args.right + first_right_row_offset;

    int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

//    if (warp.thread_rank() == 0 && warp.meta_group_rank() == 1) {
//        printf("Block: [%u, %u], Warp: %u, Right rows: %u, Y shift: %d, Right start Y: %u\n",
//               ctb.group_index().x,
//               ctb.group_index().y,
//               warp.meta_group_rank(),
//               NUM_RIGHT_ROWS,
//               y_shift,
//               warp_y_right_start
//               );
//    }


    // Preload the first values from left matrix
    T thread_left_bottom = load_with_bounds_check(
        left_row,
        warp_x_left + warp.thread_rank(),
        args.matrix_size.x
    );

    T sum[NUM_RIGHT_ROWS];
    for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
        sum[r] = 0;
    }

//    if (warp.thread_rank() == 0 && warp.meta_group_rank() == 0) {
//        printf("Block: [%u, %u], Warp: %u, X: [%u, %u]\n",
//               ctb.group_index().x,
//               ctb.group_index().y,
//               warp.meta_group_rank(),
//               args.warp_right_start.x,
//               args.warp_right_end.x
//               );
//    }

    for (
        dsize_t warp_x_right = args.warp_right_start.x;
        warp_x_right < args.warp_right_end.x;
        warp_x_right += warp.size(), warp_x_left += warp.size()
        ) {

        // Load next warp_size values
        // Load 0 if out of bounds

        // Right index will always be greater than 0 as we only
        // iterate over part of the matrix
        dsize_t right_idx = warp_x_right + warp.thread_rank();

        // Left index might be out of bounds even below 0, depending on the shift
        // It is also reading warp.size() next values, as we have warp.size() values already loaded
        // from the initialization before the for loop
        int left_idx = warp_x_left + warp.thread_rank() + warp.size();

        // Load values from num_rights right matrices
        T thread_right[NUM_RIGHT_ROWS];
        for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
            // TODO: Either do bounds check or limit the for loop below
            thread_right[r] = load_with_bounds_check(
                first_right_row + r * args.matrix_size.x,
                right_idx,
                args.matrix_size.x
            );
        }


        T thread_left_top = load_with_bounds_check(left_row, left_idx, args.matrix_size.x);

        for (dsize_t i = 0; i < warp.size(); ++i) {

            for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
                // Broadcast
                auto right_val = warp.shfl(thread_right[r], i);

                // No need to mask, if either values is out of bounds the value will be 0
                // Invert the order with respect to r, as
                //  we need the shift 0, which is the min shift
                //  at last place, so we can pass it first alone
                //  then with min_shift + 1, then with min_shift + 2 etc.
                sum[r] += thread_left_bottom * right_val;

//                if (warp.thread_rank() == 0 && warp.meta_group_rank() == 0) {
//                    printf("Block: [%u, %u], Warp: %u, Row: %u Left: %f, Right: %f\n",
//                           ctb.group_index().x,
//                           ctb.group_index().y,
//                           warp.meta_group_rank(),
//                           r,
//                           thread_left_bottom,
//                           right_val
//                    );
//                }
            }

            // Shuffle does modulo srcLane automatically
            // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
            //  making it behave as one continuous buffer
            thread_left_bottom = warp.shfl(warp.thread_rank() != 0 ? thread_left_bottom : thread_left_top,
                                           warp.thread_rank() + 1);
            thread_left_top = warp.shfl_down(thread_left_top, 1);
        }
    }

//    if (warp.thread_rank() == 0 && warp.meta_group_rank() == 1) {
//        for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
//            printf("Block: [%u, %u], Warp: %u, Rows: %u, sum[%u] == %f\n",
//                   ctb.group_index().x,
//                   ctb.group_index().y,
//                   warp.meta_group_rank(),
//                   NUM_RIGHT_ROWS,
//                   r,
//                   sum[r]
//            );
//        }
//    }

    for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
        // Res contains first the results of min_shift for all threads of the block,
        // then results of min_shift + 1 for all threads of the block,
        // up to the results of min_shift + NUM_RIGHT_ROWS in warp_shuffle_impl
        if constexpr(REVERSE_OUTPUT) {
//            if (warp.thread_rank() == 0 && warp.meta_group_rank() == 3) {
//                printf("Block: [%u, %u], Warp: %u, Rows: %u, Previous res: %f, sum[%u] == %f\n",
//                       ctb.group_index().x,
//                       ctb.group_index().y,
//                       warp.meta_group_rank(),
//                       NUM_RIGHT_ROWS,
//                       res[(NUM_RIGHT_ROWS - 1 - r) * ctb.size() + ctb.thread_rank()],
//                       r,
//                       sum[r]
//                );
//            }
            res[(NUM_RIGHT_ROWS - 1 - r) * ctb.size() + ctb.thread_rank()] += sum[r];
        } else {
//            if (warp.thread_rank() == 0 && warp.meta_group_rank() == 3) {
//                printf("Block: [%u, %u], Warp: %u, Rows: %u, Previous res: %f, sum[%u] == %f\n",
//                       ctb.group_index().x,
//                       ctb.group_index().y,
//                       warp.meta_group_rank(),
//                       NUM_RIGHT_ROWS,
//                       res[r * ctb.size() + ctb.thread_rank()],
//                       r,
//                       sum[r]
//                );
//            }
            res[r * ctb.size() + ctb.thread_rank()] += sum[r];
        }
    }
}

/*
 * First NUM_RIGHT_ROWS rows will only overlap in some of the shifts
 * If we start at the 0 row of the right matrix, then that means that the
 * top of the right matrix is inside the left matrix
 *
 * As we are computing NUM_RIGHT_ROWS shifts in consecutive rows with the same
 * x coordinate, the first shift will overlap given left row and no other shift
 * overlaps anything with the left row
 *
 * Next left row is overlapped with the args.warp_right_start.y by the following shift,
 * while the first shift overlaps the left row with args.warp_right_start.y + 1
 *
 * Then the third left row is overlapped with args.warp_right_start.y by the third shift,
 * with args.warp_right_start.y + 1 by second shift and with args.warp_right_start.y + 2 by
 * first shift etc.
 *
 * If the top of the right matrix starts outside the left matrix, which can only be above the
 * left matrix, some of the steps may be skipped, for example if it is one row above,
 * the first left row is overlapped by the first shift with row args.warp_right_start.y + 1
 * and by the second shift with row args.warp_right_start.y, which is exactly the second step described above
 *
 * Similar principle, but in reverse, applies when bottom of the right matrix is inside the left matrix.
 * There the left row stays the same, but we change the number of right rows it runs against,
 * getting progressively smaller.
 *
 * These ifs should cover all possibilities up to NUM_RIGHT_ROWS
 * Because max_shift.y - min_shift.y == NUM_RIGHT_ROWS, min_shift.y + NUM_RIGHT_ROWS == max_shift.y
 *
 */
template<int NUM_RIGHT_ROWS, dsize_t MAX_NUM_RIGHT_ROWS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void startup(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    if constexpr(NUM_RIGHT_ROWS < MAX_NUM_RIGHT_ROWS) {
//        if (warp.thread_rank() == 0) {
//            printf("Block: [%u, %u], Warp: %u, Left: %d, Right start Y: %u, Min shift: %d\n",
//                   ctb.group_index().x,
//                   ctb.group_index().y,
//                   warp.meta_group_rank(),
//                   static_cast<int>(args.warp_right_start.y) + args.warp_min_shift.y + NUM_RIGHT_ROWS - 1,
//                   args.warp_right_start.y,
//                   args.warp_min_shift.y
//            );
//        }

        if (static_cast<int>(args.warp_right_start.y) + args.warp_min_shift.y + NUM_RIGHT_ROWS - 1 >= 0) {
            compute_row_group<NUM_RIGHT_ROWS, true>(
                ctb,
                warp,
                args,
                args.warp_right_start.y ,
                args.warp_min_shift.y + NUM_RIGHT_ROWS - 1,
                res
            );
        }
        startup<NUM_RIGHT_ROWS + 1, MAX_NUM_RIGHT_ROWS>(ctb, warp, args, res);
    }
}

template<int NUM_RIGHT_ROWS, dsize_t MAX_NUM_RIGHT_ROWS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void wind_down(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    if constexpr(NUM_RIGHT_ROWS > 0) {
//        if (warp.thread_rank() == 0 && warp.meta_group_rank() == 0) {
//            printf("Block: [%u, %u], Warp: %u, Left: %u, Right end Y: %u, Max shift: %d\n",
//                   ctb.group_index().x,
//                   ctb.group_index().y,
//                   warp.meta_group_rank(),
//                   args.warp_right_end.y - NUM_RIGHT_ROWS + args.warp_max_shift.y,
//                   args.warp_right_end.y,
//                   args.warp_max_shift.y
//            );
//        }

        if (args.warp_right_end.y - NUM_RIGHT_ROWS + args.warp_max_shift.y < args.matrix_size.y) {
            compute_row_group<NUM_RIGHT_ROWS, true>(
                ctb,
                warp,
                args,
                args.warp_right_end.y - NUM_RIGHT_ROWS,
                args.warp_max_shift.y,
                res + (MAX_NUM_RIGHT_ROWS - NUM_RIGHT_ROWS) * ctb.size()
            );
        }
        wind_down<NUM_RIGHT_ROWS - 1, MAX_NUM_RIGHT_ROWS>(ctb, warp, args, res);
    }
}

template<dsize_t NUM_RIGHT_ROWS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void multirow_shuffle_impl(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    startup<1, NUM_RIGHT_ROWS>(ctb, warp, args, res);

    /*
     * The startup gets us to the situation where we have the first
     * left row at max_shift (== min_shift + NUM_RIGHTS_ROW) which is
     * to be processed with all NUM_RIGHT_ROWS
     *
     * As we are always loading warp_y_right and the following (NUM_RIGHT_ROWS - 1) rows,
     * we need to stop NUM_RIGHT_ROWS before the end
     */
    int end = args.warp_right_end.y - (NUM_RIGHT_ROWS - 1);
//    if (warp.thread_rank() == 0 && warp.meta_group_rank() <= 1) {
//        printf("Block: [%u, %u], Warp: %u, Start: %u, End: %d, Warp end: %u, NUM_RIGHT_ROWS: %u\n",
//               ctb.group_index().x,
//               ctb.group_index().y,
//               warp.meta_group_rank(),
//               args.warp_right_start.y,
//               end,
//               args.warp_right_end.y,
//               NUM_RIGHT_ROWS
//        );
//    }

    for (int warp_y_right = args.warp_right_start.y; warp_y_right < end; warp_y_right += 1) {
        compute_row_group<NUM_RIGHT_ROWS, true>(
            ctb,
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            res
        );
    }

    wind_down<NUM_RIGHT_ROWS - 1, NUM_RIGHT_ROWS>(ctb, warp, args, res);

    auto first_output_offset = args.output_pos.linear_idx(args.search_size.x);
    RES* matrix = args.out;

    // TODO: Maybe just check the x axis, Y axis should be filtered out by 0 NUM_RIGHT_ROWS
    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
            auto output_offset = first_output_offset + r * args.search_size.x;
//            if (warp.thread_rank() == 0 && warp.meta_group_rank() == 3) {
//                printf("Block: [%u, %u], Warp: %u, res[%u] == %f, First output pos: [%u, %u], Output offset: %u\n",
//                       ctb.group_index().x,
//                       ctb.group_index().y,
//                       warp.meta_group_rank(),
//                       r * ctb.size() + ctb.thread_rank(),
//                       res[r * ctb.size() + ctb.thread_rank()],
//                       args.output_pos.x,
//                       args.output_pos.y,
//                       output_offset
//                );
//            }
            auto val = res[r * ctb.size() + ctb.thread_rank()];
            if constexpr(ATOMIC) {
                atomicAdd(matrix + output_offset, val);
            } else {
//                if (output_offset == 304) {
//                    printf("Block: [%u, %u], Warp: %u, matrix[%u] == %f\n",
//                           ctb.group_index().x,
//                           ctb.group_index().y,
//                           warp.meta_group_rank(),
//                           output_offset,
//                           val
//                    );
//                }
                matrix[output_offset] = val;
            }
        }
    }
}

constexpr dsize_t max_num_right_rows = 8;
template<dsize_t NUM_RIGHT_ROWS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void multirow_shuffle_impl_dispatch(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_right_rows,
    const warp_shuffle_impl_args<T, RES>& args,
    RES* __restrict__ res
) {
    if constexpr(NUM_RIGHT_ROWS == 0) {
        // Zero is valid, if the warp is completely outside the result matrix
    } else {
        if (NUM_RIGHT_ROWS == num_right_rows) {
            multirow_shuffle_impl<NUM_RIGHT_ROWS, ATOMIC>(
                ctb,
                warp,
                args,
                res
            );
        } else {
            multirow_shuffle_impl_dispatch<NUM_RIGHT_ROWS - 1, ATOMIC>(
                ctb,
                warp,
                num_right_rows,
                args,
                res
            );
        }
    }
}


/**
 * This kernel first computes the range which should be
 * computed by the current warp in the left and right matrices
 * and then always loads 32 values
 */
template<typename T, typename RES>
__global__ void ccn_multirow_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t max_right_rows
) {
    // Initialize by loading a warp worth of data from left matrix
    // as we will be iterating over the left matrix

    // Then broadcast from the right data in sequence from all threads
    // With each broadcast, multiply and sum with the current value from
    // left matrix and then shuffle down the used values from left matrix.
    // Then shuffle the second warp worth of data from left matrix,
    // passing the last thread the value that is shuffled out of the thread 0
    // and would be forgotten
    // basically with warp size 4, it will go
    // 0 1 2 3 0 1 2 3, then 1 2 3 0 1 2 3 x, then 2 3 0 1 2 3 x x,
    // each time broadcasting first from thread 0, then 1, then 2
    // Once we get to 0 1 2 3 x x x x, we load one warp worth of values
    // from both left and right matrices

    // If the shift computed by the current thread does not overlap with the broadcast value
    // that means it tries to read from the left matrix out of bounds and thus will read 0
    // and ignore the broadcast value
    // By shifting the values down, when it reaches the part that overlaps it will receive
    // value shifted from the previous thread

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos = dsize2_t {
        ctb.group_index().x * ctb.group_dim().x,
        (ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y) * max_right_rows
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
            dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
            dsize2_t{ctb.thread_index().x, 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift = {
            static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
            static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31 for the x axis
    //
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift = {
            static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x - 1)) - static_cast<int>(half_search_size.x),
            // max_right_rows - 1 because + max_right_rows is the min_shift of next warp
            static_cast<int>(min(last_warp_thread_out_pos.y + max_right_rows - 1, search_size.y - 1)) - static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    dsize_t warp_y_right_start = max(-warp_max_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_min_shift.y, matrix_size.y);


    RES* res = shared_memory_proxy<RES>();
    for (dsize_t i = ctb.thread_rank(); i < max_right_rows * ctb.size(); i += ctb.size()) {
        res[i] = 0;
    }
    ctb.sync();

    // Max shift might be smaller than min shift if warp is completely outside the out matrix
    // +1 because max_shift is inclusive, it is the last shift computed by this warp
    // so to get the number of shifts with both sides inclusive, we need to add 1
    auto warp_num_right_rows = static_cast<dsize_t>(max(warp_max_shift.y - warp_min_shift.y + 1, 0));

//    if (warp.thread_rank() == 0 && warp.meta_group_rank() == 1) {
//        printf("Block: [%u, %u], Warp: %u, Min shift: [%d, %d], Max shift: [%d,%d], Num right rows: %u, Y: [%u, %u]\n",
//               ctb.group_index().x,
//               ctb.group_index().y,
//               warp.meta_group_rank(),
//               warp_min_shift.x,
//               warp_min_shift.y,
//               warp_max_shift.x,
//               warp_max_shift.y,
//               warp_num_right_rows,
//               warp_y_right_start,
//               warp_y_right_end
//        );
//    }

    auto args = create_warp_shuffle_impl_args(
        left,
        right,
        out,
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );

    multirow_shuffle_impl_dispatch<max_num_right_rows, false>(
        ctb,
        warp,
        warp_num_right_rows,
        args,
        res
    );

//    ctb.sync();
//    if (ctb.thread_rank() == 0) {
//        for (dsize_t i = ctb.thread_rank(); i < max_right_rows * ctb.size(); ++i) {
//            printf("res[%u] == %f\n",
//                i,
//                res[i]
//            );
//        }
//
//    }
}

template<typename T, typename RES>
void run_ccn_multirow_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t cuda_rows_per_block,
    dsize_t max_right_rows
) {
    if (cuda_rows_per_block > 32) {
        throw std::runtime_error("Too many rows per block: "s + std::to_string(cuda_rows_per_block) + " (max 32)");
    }

    dim3 num_threads(32, cuda_rows_per_block);
    dim3 num_blocks(
            div_up(search_size.x, num_threads.x),
            div_up(search_size.y, num_threads.y * max_right_rows)
    );

    dsize_t block_size = num_threads.x * num_threads.y;

    ccn_multirow_shuffle<<<num_blocks, num_threads, block_size * max_right_rows * sizeof(RES)>>>(
            left,
            right,
            out,
            matrix_size,
            search_size,
            max_right_rows
    );
}

template void run_ccn_multirow_shuffle<int, int>(
        const int* __restrict__ left,
        const int* __restrict__ right,
        int* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block,
        dsize_t max_right_rows
);

template void run_ccn_multirow_shuffle<float, float>(
        const float* __restrict__ left,
        const float* __restrict__ right,
        float* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block,
        dsize_t max_right_rows
);

template void run_ccn_multirow_shuffle<double, double>(
        const double* __restrict__ left,
        const double* __restrict__ right,
        double* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block,
        dsize_t max_right_rows
);

}
