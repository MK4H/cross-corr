#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"

#include "row_distribution.cuh"

namespace cg = cooperative_groups;

namespace cross {

constexpr unsigned int warp_size = 32;

/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    dsize2_t warp_min_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        dsize2_t warp_min_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    )   : left(left), right(right), out(out), warp_right_start(warp_right_start),
    warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), output_pos(output_pos),
    matrix_size(matrix_size), search_size(search_size)
    {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    dsize2_t warp_min_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void warp_shuffle_impl(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args
) {
    RES sum = 0;

    for (dsize_t warp_y_right = args.warp_right_start.y; warp_y_right < args.warp_right_end.y; warp_y_right += 1) {
        // In y axis, both max and min shift are equal in the current implementation
        int warp_y_left = static_cast<int>(warp_y_right) + args.warp_min_shift.y;

        const dsize_t right_row_offset = warp_y_right * args.matrix_size.x;
        const T* left_row = args.left + warp_y_left * args.matrix_size.x;

        int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

        // Preload the first values from left matrix
        T thread_left_bottom = load_with_bounds_check(
                left_row,
                warp_x_left + warp.thread_rank(),
                args.matrix_size.x
        );

        for (
                dsize_t warp_x_right = args.warp_right_start.x;
                warp_x_right < args.warp_right_end.x;
                warp_x_right += warp.size(), warp_x_left += warp.size()
                ) {

            // Load next warp_size values
            // Load 0 if out of bounds

            // Right index will always be greater than 0 as we only
            // iterate over part of the matrix
            dsize_t right_idx = warp_x_right + warp.thread_rank();

            // Left index might be out of bounds even below 0, depending on the shift
            // It is also reading warp.size() next values, as we have warp.size() values already loaded
            // from the initialization before the for loop
            int left_idx = warp_x_left + warp.thread_rank() + warp.size();

            // Load values from num_rights right matrices
            T thread_right = load_with_bounds_check(args.right + right_row_offset, right_idx, args.matrix_size.x);

            T thread_left_top = load_with_bounds_check(left_row, left_idx, args.matrix_size.x);

            for (dsize_t i = 0; i < warp.size(); ++i) {

                // Broadcast
                auto right_val = warp.shfl(thread_right, i);

                // No need to mask, if either values is out of bounds the value will be 0
                sum += thread_left_bottom * right_val;


                // Shuffle does modulo srcLane automatically
                // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
                //  making it behave as one continuous buffer
                thread_left_bottom = warp.shfl(warp.thread_rank() != 0 ? thread_left_bottom : thread_left_top,
                                               warp.thread_rank() + 1);
                thread_left_top = warp.shfl_down(thread_left_top, 1);
            }
        }
    }

    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        auto output_offset = args.output_pos.linear_idx(args.search_size.x);
        T* matrix = args.out;
        if (ATOMIC) {
            atomicAdd(matrix + output_offset, sum);
        } else {
            matrix[output_offset] = sum;
        }
    }
}

constexpr dsize_t max_num_right_matrices = 8;
/**
 * This kernel first computes the range which should be
 * computed by the current warp in the left and right matrices
 * and then always loads 32 values
 */
template<typename T, typename RES>
__global__ void ccn_warp_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    // Initialize by loading a warp worth of data from left matrix
    // as we will be iterating over the left matrix

    // Then broadcast from the right data in sequence from all threads
    // With each broadcast, multiply and sum with the current value from
    // left matrix and then shuffle down the used values from left matrix.
    // Then shuffle the second warp worth of data from left matrix,
    // passing the last thread the value that is shuffled out of the thread 0
    // and would be forgotten
    // basically with warp size 4, it will go
    // 0 1 2 3 0 1 2 3, then 1 2 3 0 1 2 3 x, then 2 3 0 1 2 3 x x,
    // each time broadcasting first from thread 0, then 1, then 2
    // Once we get to 0 1 2 3 x x x x, we load one warp worth of values
    // from both left and right matrices

    // If the shift computed by the current thread does not overlap with the broadcast value
    // that means it tries to read from the left matrix out of bounds and thus will read 0
    // and ignore the broadcast value
    // By shifting the values down, when it reaches the part that overlaps it will receive
    // value shifted from the previous thread

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos = dsize2_t {
        ctb.group_index().x * ctb.group_dim().x,
        ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
            dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
            dsize2_t{ctb.thread_index().x, 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift = {
            static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
            static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift = {
            static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x)) - static_cast<int>(half_search_size.x),
            static_cast<int>(min(last_warp_thread_out_pos.y, search_size.y)) - static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    // All threads in a warp process the same range of rows, so warp_min_shift.y and warp_max_shift.y are the same
    dsize_t warp_y_right_start = max(-warp_min_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_max_shift.y, matrix_size.y);

    auto args = create_warp_shuffle_impl_args(
        left,
        right,
        out,
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        output_pos,
        matrix_size,
        search_size
    );

    warp_shuffle_impl<false>(warp, args);
}

template<typename T, typename RES>
void run_ccn_warp_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t cuda_rows_per_block
) {
    if (cuda_rows_per_block > 32) {
        throw std::runtime_error("Too many rows per block: "s + std::to_string(cuda_rows_per_block) + " (max 32)");
    }

    dim3 num_threads(32, cuda_rows_per_block);
    dim3 num_blocks(
            div_up(search_size.x, num_threads.x),
            div_up(search_size.y, num_threads.y)
    );

    ccn_warp_shuffle<<<num_blocks, num_threads>>>(
            left,
            right,
            out,
            matrix_size,
            search_size
    );
}

template void run_ccn_warp_shuffle<int, int>(
        const int* __restrict__ left,
        const int* __restrict__ right,
        int* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block
);

template void run_ccn_warp_shuffle<float, float>(
        const float* __restrict__ left,
        const float* __restrict__ right,
        float* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block
);

template void run_ccn_warp_shuffle<double, double>(
        const double* __restrict__ left,
        const double* __restrict__ right,
        double* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block
);

}
