#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"
#include "shared_mem.cuh"

#include "row_distribution.cuh"
#include "warp_size.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

constexpr dsize_t max_num_right_rows = 4;
constexpr dsize_t max_num_right_mats = 4;

/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    vec2<int> warp_max_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        vec2<int> warp_max_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    ) : left(left), right(right), out(out), warp_right_start(warp_right_start),
        warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), warp_max_shift(warp_max_shift),
        output_pos(output_pos), matrix_size(matrix_size), search_size(search_size) {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    vec2<int> warp_max_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, bool REVERSE_OUTPUT, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void compute_row_group(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    dsize_t warp_y_right_start,
    int y_shift,
    RES* __restrict__ res
) {
    dsize_t warp_y_left = warp_y_right_start + y_shift;
    const T* left_row = args.left + warp_y_left * args.matrix_size.x;

    const dsize_t first_right_row_offset = warp_y_right_start * args.matrix_size.x;
    const T* first_right_row = args.right + first_right_row_offset;

    int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

    // Preload the first values from left matrix
    T thread_left_bottom = load_with_bounds_check(
        left_row,
        warp_x_left + warp.thread_rank(),
        args.matrix_size.x
    );

    T sum[NUM_RIGHT_ROWS * NUM_RIGHT_MATS];
    for (dsize_t r = 0; r < NUM_RIGHT_ROWS * NUM_RIGHT_MATS; ++r) {
        sum[r] = 0;
    }

    for (
        dsize_t warp_x_right = args.warp_right_start.x;
        warp_x_right < args.warp_right_end.x;
        warp_x_right += warp.size(), warp_x_left += warp.size()
    ) {

        // Load next warp_size values
        // Load 0 if out of bounds

        // Right index will always be greater than 0 as we only
        // iterate over part of the matrix
        dsize_t right_idx = warp_x_right + warp.thread_rank();

        // Left index might be out of bounds even below 0, depending on the shift
        // It is also reading warp.size() next values, as we have warp.size() values already loaded
        // from the initialization before the for loop
        int left_idx = warp_x_left + warp.thread_rank() + warp.size();

        T thread_right[NUM_RIGHT_ROWS * NUM_RIGHT_MATS];
        for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
            for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                thread_right[mat * NUM_RIGHT_ROWS + row] = load_with_bounds_check(
                    first_right_row + mat * args.matrix_size.area() + row * args.matrix_size.x,
                    right_idx,
                    args.matrix_size.x
                );
            }
        }

        T thread_left_top = load_with_bounds_check(left_row, left_idx, args.matrix_size.x);

        for (dsize_t i = 0; i < warp.size(); ++i) {
            for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
                for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                    // Broadcast
                    auto right_val = warp.shfl(thread_right[mat * NUM_RIGHT_ROWS + row], i);

                    sum[mat * NUM_RIGHT_ROWS + row] += thread_left_bottom * right_val;
                }
            }
            // Shuffle does modulo srcLane automatically
            // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
            //  making it behave as one continuous buffer
            thread_left_bottom = warp.shfl(
                warp.thread_rank() != 0 ? thread_left_bottom : thread_left_top,
                warp.thread_rank() + 1
            );
            thread_left_top = warp.shfl_down(thread_left_top, 1);
        }
    }

    for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
        for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
            // Res contains first the results of min_shift from all matrices for all threads of the block,
            // then results of min_shift + 1 from all matrices for all threads of the block,
            // up to the results of min_shift + NUM_RIGHT_ROWS for all matrices in warp_shuffle_impl
            // This is done to allow access to a subset of shifts
            // TODO: Try substituting everywhere if compiler is not able
            //dsize_t result_idx = row * NUM_RIGHT_MATS + mat;
            // Whereas sum contains all rows for the first matrix, then all rows for the second matrix etc.
            // This is done as that minimizes the jumps around memory
            // TODO: Try substituting everywhere if compiler is not able
            dsize_t sum_idx = mat * NUM_RIGHT_ROWS + row;

            if constexpr(REVERSE_OUTPUT) {
                res[((NUM_RIGHT_ROWS - 1 - row) * NUM_RIGHT_MATS + mat) * ctb.size() +
                    ctb.thread_rank()] += sum[sum_idx];
            } else {
                res[(row * NUM_RIGHT_MATS + mat) * ctb.size() + ctb.thread_rank()] += sum[sum_idx];
            }
        }
    }
}

/*
 * First NUM_RIGHT_ROWS rows will only overlap in some of the shifts
 * If we start at the 0 row of the right matrix, then that means that the
 * top of the right matrix is inside the left matrix
 *
 * As we are computing NUM_RIGHT_ROWS shifts in consecutive rows with the same
 * x coordinate, the first shift will overlap given left row and no other shift
 * overlaps anything with the left row
 *
 * Next left row is overlapped with the args.warp_right_start.y by the following shift,
 * while the first shift overlaps the left row with args.warp_right_start.y + 1
 *
 * Then the third left row is overlapped with args.warp_right_start.y by the third shift,
 * with args.warp_right_start.y + 1 by second shift and with args.warp_right_start.y + 2 by
 * first shift etc.
 *
 * If the top of the right matrix starts outside the left matrix, which can only be above the
 * left matrix, some of the steps may be skipped, for example if it is one row above,
 * the first left row is overlapped by the first shift with row args.warp_right_start.y + 1
 * and by the second shift with row args.warp_right_start.y, which is exactly the second step described above
 *
 * Similar principle, but in reverse, applies when bottom of the right matrix is inside the left matrix.
 * There the left row stays the same, but we change the number of right rows it runs against,
 * getting progressively smaller.
 *
 * These ifs should cover all possibilities up to NUM_RIGHT_ROWS
 * Because max_shift.y - min_shift.y == NUM_RIGHT_ROWS, min_shift.y + NUM_RIGHT_ROWS == max_shift.y
 *
 */
template<int NUM_RIGHT_ROWS, dsize_t MAX_NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void startup(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    if constexpr(NUM_RIGHT_ROWS < MAX_NUM_RIGHT_ROWS) {
        if (static_cast<int>(args.warp_right_start.y) + args.warp_min_shift.y + NUM_RIGHT_ROWS - 1 >= 0) {
            compute_row_group<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, true>(
                ctb,
                warp,
                args,
                args.warp_right_start.y,
                args.warp_min_shift.y + NUM_RIGHT_ROWS - 1,
                res
            );
        }
        startup<NUM_RIGHT_ROWS + 1, MAX_NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(ctb, warp, args, res);
    } else {
        // Silence the unused parameter warning
        (void)ctb;
        (void)warp;
        (void)args;
        (void)res;
    }
}

template<int NUM_RIGHT_ROWS, dsize_t MAX_NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void wind_down(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    if constexpr(NUM_RIGHT_ROWS > 0) {
        if (args.warp_right_end.y - NUM_RIGHT_ROWS + args.warp_max_shift.y < args.matrix_size.y) {
            compute_row_group<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, true>(
                ctb,
                warp,
                args,
                args.warp_right_end.y - NUM_RIGHT_ROWS,
                args.warp_max_shift.y,
                res + ((MAX_NUM_RIGHT_ROWS - NUM_RIGHT_ROWS) * NUM_RIGHT_MATS) * ctb.size()
            );
        }
        wind_down<NUM_RIGHT_ROWS - 1, MAX_NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(ctb, warp, args, res);
    } else {
        // Silence the unused parameter warning
        (void)ctb;
        (void)warp;
        (void)args;
        (void)res;
    }
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void multirow_multiright_shuffle_impl(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    startup<1, NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(ctb, warp, args, res);

    /*
     * The startup gets us to the situation where we have the first
     * left row at max_shift (== min_shift + NUM_RIGHTS_ROW) which is
     * to be processed with all NUM_RIGHT_ROWS
     *
     * As we are always loading warp_y_right and the following (NUM_RIGHT_ROWS - 1) rows,
     * we need to stop NUM_RIGHT_ROWS before the end
     */
    int end = args.warp_right_end.y - (NUM_RIGHT_ROWS - 1);

    for (int warp_y_right = args.warp_right_start.y; warp_y_right < end; warp_y_right += 1) {
        compute_row_group<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, true>(
            ctb,
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            res
        );
    }

    wind_down<NUM_RIGHT_ROWS - 1, NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(ctb, warp, args, res);

    auto first_output_offset = args.output_pos.linear_idx(args.search_size.x);
    RES* matrix = args.out;

    // TODO: Maybe just check the x axis, Y axis should be filtered out by 0 NUM_RIGHT_ROWS
    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
            for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                auto output_offset = first_output_offset + mat * args.search_size.area() + row * args.search_size.x;
                auto val = res[(row * NUM_RIGHT_MATS + mat) * ctb.size() + ctb.thread_rank()];
                if constexpr(ATOMIC) {
                    atomicAdd(matrix + output_offset, val);
                } else {
                    matrix[output_offset] = val;
                }
            }
        }
    }
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void multirow_multiright_shuffle_impl_mats_dispatch(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_right_mats,
    const warp_shuffle_impl_args<T, RES>& args,
    RES* __restrict__ res
) {
    if constexpr(NUM_RIGHT_MATS == 0) {
        // Silence the unused parameter warning
        (void)ctb;
        (void)warp;
        (void)num_right_mats;
        (void)args;
        (void)res;
        assert(false);
    } else {
        if (NUM_RIGHT_MATS == num_right_mats) {
            multirow_multiright_shuffle_impl<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, ATOMIC>(
                ctb,
                warp,
                args,
                res
            );
        } else {
            multirow_multiright_shuffle_impl_mats_dispatch<NUM_RIGHT_ROWS, NUM_RIGHT_MATS - 1, ATOMIC>(
                ctb,
                warp,
                num_right_mats,
                args,
                res
            );
        }
    }
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void multirow_multiright_shuffle_impl_rows_dispatch(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_right_rows,
    dsize_t num_right_mats,
    const warp_shuffle_impl_args<T, RES>& args,
    RES* __restrict__ res
) {
    if constexpr(NUM_RIGHT_ROWS == 0) {
        // Zero is valid, if the warp is completely outside the result matrix

        // Silence the unused parameter warning
        (void)ctb;
        (void)warp;
        (void)num_right_rows;
        (void)num_right_mats;
        (void)args;
        (void)res;
    } else {
        if (NUM_RIGHT_ROWS == num_right_rows) {
            multirow_multiright_shuffle_impl_mats_dispatch<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, ATOMIC>(
                ctb,
                warp,
                num_right_mats,
                args,
                res
            );
        } else {
            multirow_multiright_shuffle_impl_rows_dispatch<NUM_RIGHT_ROWS - 1, NUM_RIGHT_MATS, ATOMIC>(
                ctb,
                warp,
                num_right_rows,
                num_right_mats,
                args,
                res
            );
        }
    }
}


template<typename T, typename RES>
__global__ void ccn_multirow_multiright_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t max_right_rows,
    dsize_t right_matrices_per_thread
) {

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    dsize_t blocks_per_matrix_group = div_up(search_size.x, warp_size);
    dsize_t matrix_group_idx = ctb.group_index().x / blocks_per_matrix_group;
    dsize_t matrix_group_block_offset = ctb.group_index().x % blocks_per_matrix_group;

    dsize_t output_x_offset = matrix_group_block_offset * warp_size;
    dsize_t matrix_group_start_idx = matrix_group_idx * right_matrices_per_thread;

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos{
        output_x_offset,
        (ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y) * max_right_rows
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift{
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31 for the x axis
    //
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift{
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x - 1)) - static_cast<int>(half_search_size.x),
        // max_right_rows - 1 because + max_right_rows is the min_shift of next warp
        static_cast<int>(min(last_warp_thread_out_pos.y + max_right_rows - 1, search_size.y - 1)) -
        static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    dsize_t warp_y_right_start = max(-warp_max_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_min_shift.y, matrix_size.y);


    RES* res = shared_memory_proxy<RES>();
    for (dsize_t i = ctb.thread_rank(); i < max_right_rows * right_matrices_per_thread * ctb.size(); i += ctb.size()) {
        res[i] = 0;
    }
    ctb.sync();

    // Max shift might be smaller than min shift if warp is completely outside the out matrix
    // +1 because max_shift is inclusive, it is the last shift computed by this warp
    // so to get the number of shifts with both sides inclusive, we need to add 1
    auto warp_num_right_rows = static_cast<dsize_t>(max(warp_max_shift.y - warp_min_shift.y + 1, 0));

    dsize_t warp_num_right_matrices = min(num_right_matrices - matrix_group_start_idx, right_matrices_per_thread);


    auto args = create_warp_shuffle_impl_args(
        left,
        right + matrix_group_start_idx * matrix_size.area(),
        out + matrix_group_start_idx * search_size.area(),
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );

    multirow_multiright_shuffle_impl_rows_dispatch<max_num_right_rows, max_num_right_mats, false>(
        ctb,
        warp,
        warp_num_right_rows,
        warp_num_right_matrices,
        args,
        res
    );
}

} // END anonymous namespace

template<typename T, typename RES>
void run_ccn_multirow_multiright_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t cuda_rows_per_block,
    dsize_t max_right_rows,
    dsize_t right_matrices_per_thread
) {
    if (cuda_rows_per_block > 32) {
        throw std::runtime_error("Too many rows per block: "s + std::to_string(cuda_rows_per_block) + " (max 32)");
    }

    if (right_matrices_per_thread == 0 || right_matrices_per_thread > max_num_right_mats) {
        throw std::runtime_error("Invalid number of right matrices per thread: "s +
                                 std::to_string(right_matrices_per_thread) +
                                 " [1-"s +
                                 std::to_string(max_num_right_mats) +
                                 "]"s
        );
    }

    dim3 num_threads(warp_size, cuda_rows_per_block);

    dsize_t num_matrix_groups = div_up(num_right_matrices, right_matrices_per_thread);
    dsize_t blocks_per_matrix_group = div_up(search_size.x, num_threads.x);

    dim3 num_blocks(
        blocks_per_matrix_group * num_matrix_groups,
        div_up(search_size.y, num_threads.y * max_right_rows)
    );

    dsize_t block_size = num_threads.x * num_threads.y;

    dsize_t shared_mem_size = block_size * max_right_rows * right_matrices_per_thread * sizeof(RES);

    ccn_multirow_multiright_shuffle<<<num_blocks, num_threads, shared_mem_size>>>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        num_right_matrices,
        max_right_rows,
        right_matrices_per_thread
    );
}

template void run_ccn_multirow_multiright_shuffle<int, int>(
    const int* __restrict__ left,
    const int* __restrict__ right,
    int* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t cuda_rows_per_block,
    dsize_t max_right_rows,
    dsize_t right_matrices_per_thread
);

template void run_ccn_multirow_multiright_shuffle<float, float>(
    const float* __restrict__ left,
    const float* __restrict__ right,
    float* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t cuda_rows_per_block,
    dsize_t max_right_rows,
    dsize_t right_matrices_per_thread
);

template void run_ccn_multirow_multiright_shuffle<double, double>(
    const double* __restrict__ left,
    const double* __restrict__ right,
    double* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t cuda_rows_per_block,
    dsize_t max_right_rows,
    dsize_t right_matrices_per_thread
);

}
