#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"
#include "shared_mem.cuh"

#include "row_distribution.cuh"
#include "warp_size.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    vec2<int> warp_max_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        vec2<int> warp_max_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    ) : left(left), right(right), out(out), warp_right_start(warp_right_start),
        warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), warp_max_shift(warp_max_shift),
        output_pos(output_pos), matrix_size(matrix_size), search_size(search_size) {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    vec2<int> warp_max_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<dsize_t NUM_SHIFTS, dsize_t NUM_LEFT_ROWS, bool REVERSE_OUTPUT, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void compute_row_group(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    dsize_t warp_y_right_start,
    int y_shift,
    RES* __restrict__ res
) {
    dsize_t warp_y_left = warp_y_right_start + y_shift;
    const T* first_left_row = args.left + warp_y_left * args.matrix_size.x;

    const dsize_t first_right_row_offset = warp_y_right_start * args.matrix_size.x;
    const T* first_right_row = args.right + first_right_row_offset;

    int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

    // Preload the first values from left matrix
    T thread_left_bottom[NUM_LEFT_ROWS];
    #pragma unroll
    for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
        thread_left_bottom[l] = load_with_bounds_check(
            first_left_row + l * args.matrix_size.x,
            warp_x_left + warp.thread_rank(),
            args.matrix_size.x
        );
    }


    T sum[NUM_SHIFTS];
    #pragma unroll
    for (dsize_t s = 0; s < NUM_SHIFTS; ++s) {
        sum[s] = 0;
    }

    for (
        dsize_t warp_x_right = args.warp_right_start.x;
        warp_x_right < args.warp_right_end.x;
        warp_x_right += warp.size(), warp_x_left += warp.size()
        ) {

        // Load next warp_size values
        // Load 0 if out of bounds

        // Right index will always be greater than 0 as we only
        // iterate over part of the matrix
        dsize_t right_idx = warp_x_right + warp.thread_rank();

        // Left index might be out of bounds even below 0, depending on the shift
        // It is also reading warp.size() next values, as we have warp.size() values already loaded
        // from the initialization before the for loop
        int left_idx = warp_x_left + warp.thread_rank() + warp.size();

        // We need this many right values because first left row
        // is computed with rows 0 to NUM_SHIFTS - 1, second left row is computed
        // with rows 1 to NUM_SHIFTS, third left row with 2 to NUM_SHIFTS + 1
        constexpr dsize_t NUM_RIGHT_ROWS = NUM_SHIFTS + NUM_LEFT_ROWS - 1;
        // Load values from num_rights right matrices
        T thread_right[NUM_RIGHT_ROWS];
        #pragma unroll
        for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
            // TODO: Either do bounds check or limit the for loop below
            thread_right[r] = load_with_bounds_check(
                first_right_row + r * args.matrix_size.x,
                right_idx,
                args.matrix_size.x
            );
        }


        T thread_left_top[NUM_LEFT_ROWS];
        #pragma unroll
        for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
            thread_left_top[l] = load_with_bounds_check(
                first_left_row + l * args.matrix_size.x,
                left_idx,
                args.matrix_size.x
            );
        }

        // TODO: Maybe pragma unroll?
        for (dsize_t i = 0; i < warp.size(); ++i) {
            #pragma unroll
            for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
                // Broadcast
                auto right_val = warp.shfl(thread_right[r], i);

                #pragma unroll
                for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
                    // Some combinations are not valid, as described by the NUM_RIGHT_ROWS
                    // variable comment.
                    // left row 0 is computed with right rows 0 to NUM_SHIFTS - 1
                    // left row 1 is computed with right rows 1 to NUM_SHIFTS
                    // left row 2 is computed with right rows 2 to NUM_SHIFTS + 1
                    // TODO: Try if using break or continue can still be unrolled
                    if (l <= r && r < NUM_SHIFTS + l) {
                        sum[r - l] += thread_left_bottom[l] * right_val;
                    }
                }
            }

            #pragma unroll
            for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {

                // This if cannot be changed into ternary operator
                // as nvcc fails to optimize the two arrays into registers
                // and instead puts them into local memory when ternary operator
                // is used
                T bottom_shift_val;
                if (warp.thread_rank() != 0) {
                    bottom_shift_val = thread_left_bottom[l];
                } else {
                    // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
                    //  making it behave as one continuous buffer
                    bottom_shift_val = thread_left_top[l];
                }
                // Shuffle does modulo srcLane automatically
                thread_left_bottom[l] = warp.shfl(bottom_shift_val, warp.thread_rank() + 1);

                thread_left_top[l] = warp.shfl_down(thread_left_top[l], 1);
            }
        }
    }

    #pragma unroll
    for (dsize_t s = 0; s < NUM_SHIFTS; ++s) {
        // Res contains first the results of min_shift for all threads of the block,
        // then results of min_shift + 1 for all threads of the block,
        // up to the results of min_shift + NUM_RIGHT_ROWS in warp_shuffle_impl
        if constexpr(REVERSE_OUTPUT) {
            res[(NUM_SHIFTS - 1 - s) * ctb.size() + ctb.thread_rank()] += sum[s];
        } else {
            res[s * ctb.size() + ctb.thread_rank()] += sum[s];
        }
    }
}

/*
 * First NUM_RIGHT_ROWS rows will only overlap in some of the shifts
 * If we start at the 0 row of the right matrix, then that means that the
 * top of the right matrix is inside the left matrix
 *
 * As we are computing NUM_RIGHT_ROWS shifts in consecutive rows with the same
 * x coordinate, the first shift will overlap given left row and no other shift
 * overlaps anything with the left row
 *
 * Next left row is overlapped with the args.warp_right_start.y by the following shift,
 * while the first shift overlaps the left row with args.warp_right_start.y + 1
 *
 * Then the third left row is overlapped with args.warp_right_start.y by the third shift,
 * with args.warp_right_start.y + 1 by second shift and with args.warp_right_start.y + 2 by
 * first shift etc.
 *
 * If the top of the right matrix starts outside the left matrix, which can only be above the
 * left matrix, some of the steps may be skipped, for example if it is one row above,
 * the first left row is overlapped by the first shift with row args.warp_right_start.y + 1
 * and by the second shift with row args.warp_right_start.y, which is exactly the second step described above
 *
 * Similar principle, but in reverse, applies when bottom of the right matrix is inside the left matrix.
 * There the left row stays the same, but we change the number of right rows it runs against,
 * getting progressively smaller.
 *
 * These ifs should cover all possibilities up to NUM_RIGHT_ROWS
 * Because max_shift.y - min_shift.y == NUM_RIGHT_ROWS, min_shift.y + NUM_RIGHT_ROWS == max_shift.y
 *
 */
template<int NUM_THREAD_SHIFTS, dsize_t MAX_NUM_THREAD_SHIFTS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void startup(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    if constexpr(NUM_THREAD_SHIFTS < MAX_NUM_THREAD_SHIFTS) {
        if (static_cast<int>(args.warp_right_start.y) + args.warp_min_shift.y + NUM_THREAD_SHIFTS - 1 >= 0) {
            compute_row_group<NUM_THREAD_SHIFTS, 1, true>(
                ctb,
                warp,
                args,
                args.warp_right_start.y,
                args.warp_min_shift.y + NUM_THREAD_SHIFTS - 1,
                res
            );
        }
        startup<NUM_THREAD_SHIFTS + 1, MAX_NUM_THREAD_SHIFTS>(ctb, warp, args, res);
    } else {
        // Silence the unused parameter warning
        (void)ctb;
        (void)warp;
        (void)args;
        (void)res;
    }
}

template<int NUM_THREAD_SHIFTS, dsize_t MAX_NUM_THREAD_SHIFTS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void wind_down(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    if constexpr(NUM_THREAD_SHIFTS > 0) {
        if (args.warp_right_end.y - NUM_THREAD_SHIFTS + args.warp_max_shift.y < args.matrix_size.y) {
            compute_row_group<NUM_THREAD_SHIFTS, 1, true>(
                ctb,
                warp,
                args,
                args.warp_right_end.y - NUM_THREAD_SHIFTS,
                args.warp_max_shift.y,
                res + (MAX_NUM_THREAD_SHIFTS - NUM_THREAD_SHIFTS) * ctb.size()
            );
        }
        wind_down<NUM_THREAD_SHIFTS - 1, MAX_NUM_THREAD_SHIFTS>(ctb, warp, args, res);
    } else {
        // Silence the unused parameter warning
        (void)ctb;
        (void)warp;
        (void)args;
        (void)res;
    }
}

template<dsize_t NUM_THREAD_SHIFTS, dsize_t MAX_LEFT_ROWS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void multileft_shuffle_impl(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES* __restrict__ res
) {
    startup<1, NUM_THREAD_SHIFTS>(ctb, warp, args, res);

    /*
     * The startup gets us to the situation where we have the first
     * left row at max_shift (== min_shift + NUM_RIGHTS_ROW) which is
     * to be processed with all NUM_RIGHT_ROWS
     * As we are always loading warp_y_right and the following (NUM_THREAD_SHIFTS + MAX_LEFT_ROWS - 1) rows,
     * we need to stop NUM_THREAD_SHIFTS + MAX_LEFT_ROWS - 1 before the end
     */
    int multileft_end = args.warp_right_end.y - (NUM_THREAD_SHIFTS + MAX_LEFT_ROWS - 1);
    int warp_y_right = args.warp_right_start.y;
    for (; warp_y_right < multileft_end; warp_y_right += MAX_LEFT_ROWS) {
        compute_row_group<NUM_THREAD_SHIFTS, MAX_LEFT_ROWS, true>(
            ctb,
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            res
        );
    }

    /*
     * Finish the possible MAX_LEFT_ROWS - 1 left rows left over before the original wind-down
     * As we are always loading warp_y_right and the following (NUM_THREAD_SHIFTS - 1) rows,
     * we need to stop NUM_THREAD_SHIFTS before the end
     * TODO: Try template generated if tree that will use just one call with the correct number of left rows
     */
    int total_end = args.warp_right_end.y - (NUM_THREAD_SHIFTS - 1);
    for (; warp_y_right < total_end; warp_y_right += 1) {
        compute_row_group<NUM_THREAD_SHIFTS, 1, true>(
            ctb,
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            res
        );
    }

    wind_down<NUM_THREAD_SHIFTS - 1, NUM_THREAD_SHIFTS>(ctb, warp, args, res);

    auto first_output_offset = args.output_pos.linear_idx(args.search_size.x);
    RES* matrix = args.out;

    // TODO: Maybe just check the x axis, Y axis should be filtered out by 0 NUM_RIGHT_ROWS
    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        #pragma unroll
        for (dsize_t s = 0; s < NUM_THREAD_SHIFTS; ++s) {
            auto output_offset = first_output_offset + s * args.search_size.x;
            auto val = res[s * ctb.size() + ctb.thread_rank()];
            if constexpr(ATOMIC) {
                atomicAdd(matrix + output_offset, val);
            } else {
                matrix[output_offset] = val;
            }
        }
    }
}

constexpr dsize_t max_num_thread_shifts = 8;

template<dsize_t NUM_THREAD_SHIFTS, dsize_t MAX_LEFT_ROWS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void multileft_shuffle_impl_dispatch(
    const cg::thread_block& ctb,
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_thread_shifts,
    const warp_shuffle_impl_args<T, RES>& args,
    RES* __restrict__ res
) {
    if constexpr(NUM_THREAD_SHIFTS == 0) {
        // Zero is valid, if the warp is completely outside the result matrix

        // Silence the unused parameter warning
        (void)ctb;
        (void)warp;
        (void)num_thread_shifts;
        (void)args;
        (void)res;
    } else {
        if (NUM_THREAD_SHIFTS == num_thread_shifts) {
            multileft_shuffle_impl<NUM_THREAD_SHIFTS, MAX_LEFT_ROWS, ATOMIC>(
                ctb,
                warp,
                args,
                res
            );
        } else {
            multileft_shuffle_impl_dispatch<NUM_THREAD_SHIFTS - 1, MAX_LEFT_ROWS, ATOMIC>(
                ctb,
                warp,
                num_thread_shifts,
                args,
                res
            );
        }
    }
}


/**
 * This kernel first computes the range which should be
 * computed by the current warp in the left and right matrices
 * and then always loads 32 values
 */
template<dsize_t MAX_LEFT_ROWS, typename T, typename RES>
__global__ void ccn_multileft_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t max_shifts_per_thread
) {
    // Initialize by loading a warp worth of data from left matrix
    // as we will be iterating over the left matrix

    // Then broadcast from the right data in sequence from all threads
    // With each broadcast, multiply and sum with the current value from
    // left matrix and then shuffle down the used values from left matrix.
    // Then shuffle the second warp worth of data from left matrix,
    // passing the last thread the value that is shuffled out of the thread 0
    // and would be forgotten
    // basically with warp size 4, it will go
    // 0 1 2 3 0 1 2 3, then 1 2 3 0 1 2 3 x, then 2 3 0 1 2 3 x x,
    // each time broadcasting first from thread 0, then 1, then 2
    // Once we get to 0 1 2 3 x x x x, we load one warp worth of values
    // from both left and right matrices

    // If the shift computed by the current thread does not overlap with the broadcast value
    // that means it tries to read from the left matrix out of bounds and thus will read 0
    // and ignore the broadcast value
    // By shifting the values down, when it reaches the part that overlaps it will receive
    // value shifted from the previous thread

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos{
        ctb.group_index().x * ctb.group_dim().x,
        (ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y) * max_shifts_per_thread
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift = {
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31 for the x axis
    //
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift = {
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x - 1)) -
        static_cast<int>(half_search_size.x),
        // max_right_rows - 1 because + max_right_rows is the min_shift of next warp
        static_cast<int>(min(last_warp_thread_out_pos.y + max_shifts_per_thread - 1, search_size.y - 1)) -
        static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    dsize_t warp_y_right_start = max(-warp_max_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_min_shift.y, matrix_size.y);


    RES* res = shared_memory_proxy<RES>();
    for (dsize_t i = ctb.thread_rank(); i < max_shifts_per_thread * ctb.size(); i += ctb.size()) {
        res[i] = 0;
    }
    ctb.sync();

    // Max shift might be smaller than min shift if warp is completely outside the out matrix
    // +1 because max_shift is inclusive, it is the last shift computed by this warp
    // so to get the number of shifts with both sides inclusive, we need to add 1
    auto num_thread_shifts = static_cast<dsize_t>(max(warp_max_shift.y - warp_min_shift.y + 1, 0));

    auto args = create_warp_shuffle_impl_args(
        left,
        right,
        out,
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );

    multileft_shuffle_impl_dispatch<max_num_thread_shifts, MAX_LEFT_ROWS, false>(
        ctb,
        warp,
        num_thread_shifts,
        args,
        res
    );
}

constexpr dsize_t left_rows_limit = 4;

template<dsize_t MAX_LEFT_ROWS, typename T, typename RES>
__host__ void ccn_multileft_shuffle_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t cuda_rows_per_block,
    dsize_t max_shifts_per_thread,
    dsize_t max_left_rows
) {
    if constexpr(MAX_LEFT_ROWS > 0) {
        if (MAX_LEFT_ROWS == max_left_rows) {
            dim3 num_threads(warp_size, cuda_rows_per_block);
            dim3 num_blocks(
                div_up(search_size.x, num_threads.x),
                div_up(search_size.y, num_threads.y * max_shifts_per_thread)
            );

            dsize_t block_size = num_threads.x * num_threads.y;
            dsize_t shared_mem_size = block_size * max_shifts_per_thread * sizeof(RES);

            ccn_multileft_shuffle<MAX_LEFT_ROWS><<<num_blocks, num_threads, shared_mem_size>>>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                max_shifts_per_thread
            );
        } else {
            ccn_multileft_shuffle_dispatch<MAX_LEFT_ROWS - 1>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                cuda_rows_per_block,
                max_shifts_per_thread,
                max_left_rows
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)cuda_rows_per_block;
        (void)max_shifts_per_thread;
        (void)max_left_rows;
        assert(false);
    }
}

} // END anonymous namespace

template<typename T, typename RES>
void run_ccn_multileft_shuffle(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t cuda_rows_per_block,
    dsize_t max_shifts_per_thread,
    dsize_t max_left_rows
) {
    if (cuda_rows_per_block > 32) {
        throw std::runtime_error("Too many rows per block: "s + std::to_string(cuda_rows_per_block) + " (max 32)");
    }

    if (max_shifts_per_thread > max_num_thread_shifts) {
        throw std::runtime_error(
            "Too many shifts per thread: "s +
            std::to_string(max_shifts_per_thread) +
            "(max "s + std::to_string(max_num_thread_shifts) +
            ")"s
        );
    }

    if (max_left_rows > left_rows_limit) {
        throw std::runtime_error(
            "Too many left rows per iteration: "s +
            std::to_string(max_left_rows) +
            "(max "s + std::to_string(left_rows_limit) +
            ")"s
        );
    }

    ccn_multileft_shuffle_dispatch<left_rows_limit>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        cuda_rows_per_block,
        max_shifts_per_thread,
        max_left_rows
    );
}

template void run_ccn_multileft_shuffle<int, int>(
        const int* __restrict__ left,
        const int* __restrict__ right,
        int* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block,
        dsize_t max_shifts_per_thread,
        dsize_t max_left_rows
);

template void run_ccn_multileft_shuffle<float, float>(
        const float* __restrict__ left,
        const float* __restrict__ right,
        float* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block,
        dsize_t max_shifts_per_thread,
        dsize_t max_left_rows
);

template void run_ccn_multileft_shuffle<double, double>(
        const double* __restrict__ left,
        const double* __restrict__ right,
        double* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t cuda_rows_per_block,
        dsize_t max_shifts_per_thread,
        dsize_t max_left_rows
);

}
